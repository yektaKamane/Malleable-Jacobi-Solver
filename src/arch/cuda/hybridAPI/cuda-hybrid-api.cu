/* 
 * cuda-hybrid-api.cu
 *
 * by Lukasz Wesolowski
 * 04.01.2008
 *
 * an interface for execution on the GPU
 *
 * description: 
 * -user enqueues one or more work requests to the work
 * request queue (wrQueue) to be executed on the GPU
 * - a converse function (gpuProgressFn) executes periodically to
 * offload work requests to the GPU one at a time
 *
 */

#include "wrqueue.h"
#include "cuda-hybrid-api.h"

workRequestQueue *wrQueue = NULL; 

/*
  TO DO
  stream 1 - kernel execution
  stream 2 - memory setup
  stream 3 - memory copies
*/

/* setupMemory
   set up memory on the gpu for this kernel's execution */
void setupMemory(workRequest *wr) {
  hipMalloc((void **)&(wr->readWriteDevicePtr), wr->readWriteLen);
  hipMalloc((void **)&(wr->readOnlyDevicePtr), wr->readOnlyLen); 
  hipMalloc((void **)&(wr->writeOnlyDevicePtr), wr->writeOnlyLen);
  
  hipMemcpy(wr->readWriteDevicePtr, wr->readWriteHostPtr, wr->readWriteLen, 
		  hipMemcpyHostToDevice); 
  hipMemcpy(wr->readOnlyDevicePtr, wr->readOnlyHostPtr, wr->readOnlyLen, 
		  hipMemcpyHostToDevice); 
  
} 

/* cleanupMemory
   free memory no longer needed on the gpu */ 
void cleanupMemory(workRequest *wr) {
  
  hipMemcpy(wr->readWriteHostPtr, wr->readWriteDevicePtr, wr->readWriteLen, hipMemcpyDeviceToHost); 
  hipMemcpy(wr->writeOnlyHostPtr, wr->writeOnlyDevicePtr, wr->writeOnlyLen, hipMemcpyDeviceToHost); 
  

  hipFree(wr->readWriteDevicePtr); 
  hipFree(wr->readOnlyDevicePtr); 
  hipFree(wr->writeOnlyDevicePtr);

}

/* kernelSelect
   a switch statement defined by the user to allow the library to execute
   the correct kernel */ 
void kernelSelect(workRequest *wr);

/* initHybridAPI
   initializes the work request queue
*/
void initHybridAPI() {
  initWRqueue(&wrQueue); 
}

/* gpuProgressFn
   called periodically to check if the current kernel has completed,
   and invoke subsequent kernel */
void gpuProgressFn() {
  if (wrQueue == NULL) {
    return; 
  }

  while (!isEmpty(wrQueue)) {
    workRequest *wr = head(wrQueue); 
    
    if (wr->executing == 0) {
      setupMemory(wr); 
      kernelSelect(wr); 
      // hipEventRecord(wr->completionEvent, 0);
      wr->executing = 1; 
      return; 
    }  
    // else if (hipEventQuery(wr->completionEvent) == hipSuccess ) {      
    else if (hipStreamQuery(0) == hipSuccess ) {      
      cleanupMemory(wr);
      dequeue(wrQueue);
      wr->callbackFn();
    }
      
  }
}

/* exitHybridAPI
   cleans up and deletes memory allocated for the queue
*/
void exitHybridAPI() {
  deleteWRqueue(wrQueue); 
}
