#include "hip/hip_runtime.h"
/* 
 * cuda-hybrid-api.cu
 *
 * by Lukasz Wesolowski
 * 04.01.2008
 *
 * an interface for execution on the GPU
 *
 * description: 
 * -user enqueues one or more work requests to the work
 * request queue (wrQueue) to be executed on the GPU
 * - a converse function (gpuProgressFn) executes periodically to
 * offload work requests to the GPU one at a time
 *
 */

#include "wrqueue.h"
#include "cuda-hybrid-api.h"
#include "stdio.h"


/* A function in ck.C which casts the void * to a CkCallback object
 *  and executes the callback 
 */ 
extern void CUDACallbackManager(void * fn); 

/* initial size of host/device buffer arrays - dynamically expanded by
 *  the runtime system if needed
 */ 
#define NUM_BUFFERS 100

//#define GPU_DEBUG


/* a flag which tells the system to record the time for invocation and
 *  completion of GPU events: memory allocation, transfer and
 *  kernel execution
 */  
// #define GPUTIME

/* work request queue */
workRequestQueue *wrQueue = NULL; 

/* The runtime system keeps track of all allocated buffers on the GPU.
 * The following arrays contain pointers to host (CPU) data and the
 * corresponding data on the device (GPU). 
 */ 

/* host buffers  */ 
void **hostBuffers = NULL; 

/* device buffers */
void **devBuffers = NULL; 


#ifdef GPUTIME
#include <time.h>
/* event types */
#define MEMORY_ALLOCATION     1
#define CPU_GPU_TRANSFER      2
#define GPU_CPU_TRANSFER      3
#define MEMORY_FREE           4
#define KERNEL_EXECUTION      5

typedef struct gpuEventTimer {
  clock_t startTime; 
  clock_t endTime; 
  int eventType;
  int ID; 
} gpuEventTimer; 

gpuEventTimer gpuEvents[QUEUE_SIZE_INIT * 3]; 
int timeIndex = 0; 

#endif

/* There are separate CUDA streams for kernel execution, data transfer
 *  into the device, and data transfer out. This allows prefetching of
 *  data for a subsequent kernel while the previous kernel is
 *  executing and transferring data out of the device. 
 */
hipStream_t kernel_stream; 
hipStream_t data_in_stream;
hipStream_t data_out_stream; 

/* setupData
 *  sets up data on the gpu before kernel execution 
 */
void setupData(workRequest *wr) {
  int returnVal;
  dataInfo *bufferInfo = wr->bufferInfo; 

  if (bufferInfo != NULL) {
    for (int i=0; i<wr->nBuffers; i++) {
      int index = bufferInfo[i].bufferID; 
      int size = bufferInfo[i].size; 
      hostBuffers[index] = bufferInfo[i].hostBuffer; 
      
      /* allocate if the buffer for the corresponding index is NULL */
      if (devBuffers[index] == NULL) {
#ifdef GPU_DEBUG
	printf("buffer %d allocated\n", index); 
#endif
#ifdef GPUTIME
	gpuEvents[timeIndex].startTime = clock(); 
	gpuEvents[timeIndex].eventType = MEMORY_ALLOCATION; 
	gpuEvents[timeIndex].ID = index; 
#endif
	returnVal = hipMalloc((void **) &devBuffers[index], size); 
#ifdef GPUTIME
	gpuEvents[timeIndex].endTime = clock(); 
	timeIndex++;
#endif
#ifdef GPU_DEBUG
	printf("hipMalloc returned %d\n", returnVal); 
#endif
      }
      
      if (bufferInfo[i].transferToDevice) {
#ifdef GPU_DEBUG
	printf("transferToDevice bufId: %d\n", index); 
#endif

	hipMemcpyAsync(devBuffers[index], hostBuffers[index], size, 
			hipMemcpyHostToDevice, data_in_stream);
      }
    }
  }
} 

/* copybackData
 *  transfer data from the GPU to the CPU after a work request is done 
 */ 
void copybackData(workRequest *wr) {
  dataInfo *bufferInfo = wr->bufferInfo; 

  if (bufferInfo != NULL) {
    int nBuffers = wr->nBuffers; 
    
    for (int i=0; i<nBuffers; i++) {
      int index = bufferInfo[i].bufferID; 
      int size = bufferInfo[i].size; 
      
      if (bufferInfo[i].transferFromDevice) {
#ifdef GPU_DEBUG
	printf("transferFromDevice: %d\n", index); 
#endif

	hipMemcpyAsync(hostBuffers[index], devBuffers[index], size,
			hipMemcpyDeviceToHost, data_out_stream);
      }
    }     
  }
}

/* frees GPU memory for buffers specified by the user; also frees the
 *  work request's bufferInfo array
 */
void freeMemory(workRequest *wr) {
  dataInfo *bufferInfo = wr->bufferInfo;   
  int nBuffers = wr->nBuffers; 
  if (bufferInfo != NULL) {
    for (int i=0; i<nBuffers; i++) {    
      int index = bufferInfo[i].bufferID; 
      if (bufferInfo[i].freeBuffer) {
#ifdef GPU_DEBUG
	printf("buffer %d freed\n", index);
#endif 
	hipFree(devBuffers[index]); 
	devBuffers[index] = NULL; 
      }
    }
    free(bufferInfo); 
  }
}

/* kernelSelect
 * a switch statement defined by the user to allow the library to execute
 * the correct kernel 
 */ 
void kernelSelect(workRequest *wr);

/* initHybridAPI
 *   initializes the work request queue, host/device buffer pointer
 *   arrays, and CUDA streams
 */
void initHybridAPI() {
  initWRqueue(&wrQueue);

  /* allocate host/device buffers array */
  hostBuffers = (void **) malloc(NUM_BUFFERS * sizeof(void *)); 
  devBuffers = (void **) malloc(NUM_BUFFERS * sizeof(void *)); 

  /* initialize device array to NULL */ 
  for (int i=0; i<NUM_BUFFERS; i++) {
    devBuffers[i] = NULL; 
  }
  
  hipStreamCreate(&kernel_stream); 
  hipStreamCreate(&data_in_stream); 
  hipStreamCreate(&data_out_stream); 

}

/* gpuProgressFn
 *  called periodically to monitor work request progress, and perform
 *  the prefetch of data for a subsequent work request
 */
void gpuProgressFn() {

  if (wrQueue == NULL) {
    return; 
  }

  while (!isEmpty(wrQueue)) {
    int returnVal; 
    workRequest *wr = head(wrQueue); 
    workRequest *second = next(wrQueue); 
    
    if (wr->state == QUEUED) {
      setupData(wr); 
      wr->state = TRANSFERRING_IN; 
      return; 
    }  
    else if (wr->state == TRANSFERRING_IN) {
      if ((returnVal = hipStreamQuery(data_in_stream)) == hipSuccess) {

#ifdef GPUTIME
	gpuEvents[timeIndex].startTime = clock(); 
	gpuEvents[timeIndex].eventType = KERNEL_EXECUTION; 
	gpuEvents[timeIndex].ID = wr->id; 
#endif
	kernelSelect(wr); 
	wr->state = EXECUTING; 
      }
#ifdef GPU_DEBUG
      printf("Querying memory stream returned: %d\n", returnVal);
#endif  
    }
    else if (wr->state == EXECUTING) {
      if ((returnVal = hipStreamQuery(kernel_stream)) == hipSuccess) {
#ifdef GPUTIME
	gpuEvents[timeIndex].endTime = clock(); 
	timeIndex++; 
#endif
        copybackData(wr);
	wr->state = TRANSFERRING_OUT;
      }
#ifdef GPU_DEBUG
      printf("Querying kernel completion returned: %d \n", returnVal);
#endif  

      /* prefetch data for the subsequent kernel */
      if (second != NULL && second->state == QUEUED) {
	setupData(second); 
	second->state = TRANSFERRING_IN; 
	return; 
      }
    }
    else if (wr->state == TRANSFERRING_OUT) {
      if (hipStreamQuery(data_out_stream) == hipSuccess) {
	freeMemory(wr); 
	dequeue(wrQueue);
	CUDACallbackManager(wr->callbackFn);
      }
    }
#ifdef GPU_DEBUG
    else {
      printf("Error: unrecognized state\n"); 
      return; 
    }
#endif
  }
}

/* exitHybridAPI
 *  cleans up and deletes memory allocated for the queue and the CUDA streams
 */
void exitHybridAPI() {
  deleteWRqueue(wrQueue); 
  hipStreamDestroy(kernel_stream); 
  hipStreamDestroy(data_in_stream); 
  hipStreamDestroy(data_out_stream); 

#ifdef GPU_DEBUG
  for (int i=0; i<timeIndex; i++) {
    switch (gpuEvents[timeIndex].eventType) {
    case (MEMORY_ALLOCATION) :
      printf("Buffer %d allocation ", gpuEvents[timeIndex].ID); 
      break; 
    case (KERNEL_EXECUTION) :
      printf("Kernel %d execution ", gpuEvents[timeIndex].ID); 
      break;
    default:
      printf("Error, invalid timer identifier\n"); 
    }
    printf("%d:%d\n", gpuEvents[timeIndex].startTime, gpuEvents[timeIndex].endTime); 
  }
#endif
}






