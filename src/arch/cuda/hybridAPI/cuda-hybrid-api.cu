/* 
 * cuda-hybrid-api.cu
 *
 * by Lukasz Wesolowski
 * 04.01.2008
 *
 * an interface for execution on the GPU
 *
 * description: 
 * -user enqueues one or more work requests to the work
 * request queue (wrQueue) to be executed on the GPU
 * - a converse function (gpuProgressFn) executes periodically to
 * offload work requests to the GPU one at a time
 *
 */

#include "wrqueue.h"
#include "cuda-hybrid-api.h"
#include "stdio.h"
#include <cutil.h>

/* A function in ck.C which casts the void * to a CkCallback object
 *  and executes the callback 
 */ 
extern void CUDACallbackManager(void * fn); 
extern int CmiMyPe();

/* initial size of the user-addressed portion of host/device buffer
 * arrays; the system-addressed portion of host/device buffer arrays
 * (used when there is no need to share buffers between work requests)
 * will be equivalant in size.  
 */ 
#define NUM_BUFFERS 256
#define MAX_PINNED_REQ 64  
#define MAX_DELAYED_FREE_REQS 64  

/* a flag which tells the system to record the time for invocation and
 *  completion of GPU events: memory allocation, transfer and
 *  kernel execution
 */  
//#define GPU_PROFILE
//#define GPU_DEBUG
//#define GPU_TRACE
//#define _DEBUG

/* work request queue */
workRequestQueue *wrQueue = NULL; 

/* pending page-locked memory allocation requests */
unsigned int pinnedMemQueueIndex = 0; 
pinnedMemReq pinnedMemQueue[MAX_PINNED_REQ];

unsigned int currentDfr = 0;
void *delayedFreeReqs[MAX_DELAYED_FREE_REQS];

#ifdef GPU_MEMPOOL
#define GPU_MEMPOOL_NUM_SLOTS 15

CkVec<BufferPool> memPoolFreeBufs;
CkVec<int> memPoolBoundaries;
//int memPoolBoundaries[GPU_MEMPOOL_NUM_SLOTS];
#endif

/* The runtime system keeps track of all allocated buffers on the GPU.
 * The following arrays contain pointers to host (CPU) data and the
 * corresponding data on the device (GPU). 
 */ 

/* host buffers  */ 
void **hostBuffers = NULL; 

/* device buffers */
void **devBuffers = NULL; 

/* used to assign bufferIDs automatically by the system if the user 
   specifies an invalid bufferID */
unsigned int nextBuffer; 

unsigned int timerHandle; 

#ifdef GPU_PROFILE

/* event types */
#define DATA_SETUP          1            
#define KERNEL_EXECUTION    2
#define DATA_CLEANUP        3

typedef struct gpuEventTimer {
  float startTime; 
  float endTime; 
  int eventType;
  int ID; 
#ifdef GPU_TRACE
  int stage; 
  double cmistartTime; 
  double cmiendTime; 
#endif
} gpuEventTimer; 

gpuEventTimer gpuEvents[QUEUE_SIZE_INIT * 3]; 
unsigned int timeIndex = 0; 
unsigned int runningKernelIndex = 0; 
unsigned int dataSetupIndex = 0; 
unsigned int dataCleanupIndex = 0; 

#if defined GPU_TRACE || defined GPU_INSTRUMENT_WRS
extern "C" double CmiWallTimer(); 
#endif

#ifdef GPU_TRACE
extern "C" int traceRegisterUserEvent(const char*x, int e);
extern "C" void traceUserBracketEvent(int e, double beginT, double endT);

#define GPU_MEM_SETUP 8800
#define GPU_KERNEL_EXEC 8801
#define GPU_MEM_CLEANUP 8802

#endif

#endif

#ifdef GPU_INSTRUMENT_WRS
CkVec<CkVec<CkVec<RequestTimeInfo> > > avgTimes;
bool initialized_instrument;
bool initializedInstrument();
#endif

/* There are separate CUDA streams for kernel execution, data transfer
 *  into the device, and data transfer out. This allows prefetching of
 *  data for a subsequent kernel while the previous kernel is
 *  executing and transferring data out of the device. 
 */
hipStream_t kernel_stream; 
hipStream_t data_in_stream;
hipStream_t data_out_stream; 

/* pinnedMallocHost
 *
 * schedules a pinned memory allocation so that it does not impede
 * concurrent asynchronous execution 
 *
 */
void pinnedMallocHost(pinnedMemReq *reqs) {

  if ( (hipStreamQuery(kernel_stream) == hipSuccess) &&
       (hipStreamQuery(data_in_stream) == hipSuccess) &&
       (hipStreamQuery(data_out_stream) == hipSuccess) ) {    



    for (int i=0; i<reqs->nBuffers; i++) {
      CUDA_SAFE_CALL_NO_SYNC(hipHostMalloc((void **) reqs->hostPtrs[i], 
					    reqs->sizes[i])); 
    }

    free(reqs->hostPtrs);
    free(reqs->sizes);

    CUDACallbackManager(reqs->callbackFn);

  }
  else {
    pinnedMemQueue[pinnedMemQueueIndex].hostPtrs = reqs->hostPtrs;
    pinnedMemQueue[pinnedMemQueueIndex].sizes = reqs->sizes; 
    pinnedMemQueue[pinnedMemQueueIndex].callbackFn = reqs->callbackFn;     
    pinnedMemQueueIndex++;
    if (pinnedMemQueueIndex == MAX_PINNED_REQ) {
      printf("Error: pinned memory request buffer is overflowing\n"); 
    }
  }
}

void delayedFree(void *ptr){
  if(currentDfr == MAX_DELAYED_FREE_REQS){
    printf("Ran out of DFR queue space. Increase MAX_DELAYED_FREE_REQS\n");
    exit(-1);
  }
  else{
    delayedFreeReqs[currentDfr] = ptr;
  }
  currentDfr++;
}

void flushDelayedFrees(){
  for(int i = 0; i < currentDfr; i++){
    if(delayedFreeReqs[i] == NULL){
      printf("recorded NULL ptr in delayedFree()");
      exit(-1);
    }
    hipHostFree(delayedFreeReqs[i]);
  }
  currentDfr = 0; 
}

/* flushPinnedMemQueue
 *
 * executes pending pinned memory allocation requests
 *
 */
void flushPinnedMemQueue() {

  for (int i=0; i<pinnedMemQueueIndex; i++) {
    pinnedMemReq *req = &pinnedMemQueue[i]; 
    for (int j=0; j<req->nBuffers; j++) {
      CUDA_SAFE_CALL_NO_SYNC(hipHostMalloc((void **) req->hostPtrs[j], 
					    req->sizes[j])); 
    }
    free(req->hostPtrs);
    free(req->sizes);
    CUDACallbackManager(pinnedMemQueue[i].callbackFn);    
  }
  pinnedMemQueueIndex = 0; 

}

/* allocateBuffers
 *
 * allocates a work request's data on the GPU
 *
 * used to allocate memory for work request data in advance in order
 * to allow overlapping the work request's data transfer to the GPU
 * with the execution of the previous kernel; the allocation needs to
 * take place before the kernel starts executing in order to allow overlap
 *
 */

void allocateBuffers(workRequest *wr) {
  dataInfo *bufferInfo = wr->bufferInfo; 

  if (bufferInfo != NULL) {

    for (int i=0; i<wr->nBuffers; i++) {
      int index = bufferInfo[i].bufferID; 
      int size = bufferInfo[i].size; 

      // if index value is invalid, use an available ID  
      if (index < 0 || index >= NUM_BUFFERS) {
	int found = 0; 
	for (int j=nextBuffer; j<NUM_BUFFERS*2; j++) {
	  if (devBuffers[j] == NULL) {
	    index = j;
	    found = 1; 
	    break;
	  }
	}

	/* if no index was found, try to search for a value at the
	 * beginning of the system addressed space 
	 */
	
	if (!found) {
	  for (int j=NUM_BUFFERS; j<nextBuffer; j++) {
	    if (devBuffers[j] == NULL) {	
	      index = j;
	      found = 1; 
	      break;
	    }
	  }
	}

	/* if no index was found, print an error */
	if (!found) {
	  printf("Error: devBuffers is full \n");
	}

	nextBuffer = index+1; 
	if (nextBuffer == NUM_BUFFERS * 2) {
	  nextBuffer = NUM_BUFFERS; 
	}
	
	bufferInfo[i].bufferID = index; 

      }      
      
      // allocate if the buffer for the corresponding index is NULL 
      if (devBuffers[index] == NULL && size > 0) {
#ifdef GPU_PRINT_BUFFER_ALLOCATE
        double mil = 1e3;
        printf("*** ALLOCATE buffer 0x%x (%d) size %f kb\n", devBuffers[index], index, 1.0*size/mil);

#endif

        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &devBuffers[index], size));
#ifdef GPU_DEBUG
        printf("buffer %d allocated at time %.2f size: %d error string: %s\n", 
	       index, cutGetTimerValue(timerHandle), size, 
	       hipGetErrorString( hipGetLastError() ) );
#endif
      }
    }
  }
}


/* setupData
 *  copy data to the GPU before kernel execution 
 */
void setupData(workRequest *wr) {
  dataInfo *bufferInfo = wr->bufferInfo; 

  if (bufferInfo != NULL) {
    for (int i=0; i<wr->nBuffers; i++) {
      int index = bufferInfo[i].bufferID; 
      int size = bufferInfo[i].size; 
      hostBuffers[index] = bufferInfo[i].hostBuffer; 
      
      /* allocate if the buffer for the corresponding index is NULL */
      /*
      if (devBuffers[index] == NULL) {
	CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &devBuffers[index], size));
#ifdef GPU_DEBUG
	printf("buffer %d allocated %.2f\n", index,
	       cutGetTimerValue(timerHandle)); 
#endif
      }
      */
      
      if (bufferInfo[i].transferToDevice && size > 0) {
	CUDA_SAFE_CALL_NO_SYNC(hipMemcpyAsync(devBuffers[index], 
          hostBuffers[index], size, hipMemcpyHostToDevice, data_in_stream));
#ifdef GPU_DEBUG
	printf("transferToDevice bufId: %d at time %.2f size: %d " 
	       "error string: %s\n", index, cutGetTimerValue(timerHandle), 
	       size, hipGetErrorString( hipGetLastError() )); 
#endif	
	/*
	CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(devBuffers[index], 
          hostBuffers[index], size, hipMemcpyHostToDevice));
	*/

      }
    }
  }
} 

/* copybackData
 *  transfer data from the GPU to the CPU after a work request is done 
 */ 
void copybackData(workRequest *wr) {
  dataInfo *bufferInfo = wr->bufferInfo; 

  if (bufferInfo != NULL) {
    int nBuffers = wr->nBuffers; 
    
    for (int i=0; i<nBuffers; i++) {
      int index = bufferInfo[i].bufferID; 
      int size = bufferInfo[i].size; 
      
      if (bufferInfo[i].transferFromDevice && size > 0) {
#ifdef GPU_DEBUG
	printf("transferFromDevice: %d at time %.2f size: %d "
	       "error string: %s\n", index, cutGetTimerValue(timerHandle), 
	       size, hipGetErrorString( hipGetLastError() )); 
#endif
	
	CUDA_SAFE_CALL_NO_SYNC(hipMemcpyAsync(hostBuffers[index], 
          devBuffers[index], size, hipMemcpyDeviceToHost,
          data_out_stream));
	
	/*
	CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(hostBuffers[index], 
          devBuffers[index], size, hipMemcpyDeviceToHost));
	*/
      }
    }     
  }
}

/* frees GPU memory for buffers specified by the user; also frees the
 *  work request's bufferInfo array
 */
void freeMemory(workRequest *wr) {
  dataInfo *bufferInfo = wr->bufferInfo;   
  int nBuffers = wr->nBuffers; 
  if (bufferInfo != NULL) {
    for (int i=0; i<nBuffers; i++) {    
      int index = bufferInfo[i].bufferID; 
      if (bufferInfo[i].freeBuffer) {
#ifdef GPU_PRINT_BUFFER_ALLOCATE
        printf("*** FREE buffer 0x%x (%d)\n", devBuffers[index], index);
#endif

#ifdef GPU_DEBUG
        printf("buffer %d freed at time %.2f error string: %s\n", 
	       index, cutGetTimerValue(timerHandle),  
	       hipGetErrorString( hipGetLastError() ));
#endif 
        CUDA_SAFE_CALL_NO_SYNC(hipFree(devBuffers[index])); 
        devBuffers[index] = NULL; 
      }
    }
    free(bufferInfo); 
  }
}

/* 
 * a switch statement defined by the user to allow the library to execute
 * the correct kernel 
 */ 
void kernelSelect(workRequest *wr);

/* initHybridAPI
 *   initializes the work request queue, host/device buffer pointer
 *   arrays, and CUDA streams
 */
void initHybridAPI(int myPe) {

  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  hipSetDevice(myPe % deviceCount); 

  initWRqueue(&wrQueue);

  /* allocate host/device buffers array (both user and
     system-addressed) */
  hostBuffers = (void **) malloc(NUM_BUFFERS * 2 * sizeof(void *)); 
  devBuffers = (void **) malloc(NUM_BUFFERS * 2 * sizeof(void *)); 

  /* initialize device array to NULL */ 
  for (int i=0; i<NUM_BUFFERS*2; i++) {
    devBuffers[i] = NULL; 
  }
  
  CUDA_SAFE_CALL_NO_SYNC(hipStreamCreate(&kernel_stream)); 
  CUDA_SAFE_CALL_NO_SYNC(hipStreamCreate(&data_in_stream)); 
  CUDA_SAFE_CALL_NO_SYNC(hipStreamCreate(&data_out_stream)); 

#ifdef GPU_PROFILE
  CUT_SAFE_CALL(cutCreateTimer(&timerHandle));
  CUT_SAFE_CALL(cutStartTimer(timerHandle));
#endif

  nextBuffer = NUM_BUFFERS;  

#ifdef GPU_TRACE
  traceRegisterUserEvent("GPU Memory Setup", GPU_MEM_SETUP);
  traceRegisterUserEvent("GPU Kernel Execution", GPU_KERNEL_EXEC);
  traceRegisterUserEvent("GPU Memory Cleanup", GPU_MEM_CLEANUP);
#endif

#ifdef GPU_MEMPOOL
  int nslots = GPU_MEMPOOL_NUM_SLOTS;
  int *sizes;
  sizes = (int *)malloc(sizeof(int)*nslots); 

  memPoolBoundaries.reserve(GPU_MEMPOOL_NUM_SLOTS);
  memPoolBoundaries.length() = GPU_MEMPOOL_NUM_SLOTS;

  int bufSize = GPU_MEMPOOL_MIN_BUFFER_SIZE;
  for(int i = 0; i < GPU_MEMPOOL_NUM_SLOTS; i++){
    memPoolBoundaries[i] = bufSize;
    bufSize = bufSize << 1;
  }

  //1K
  sizes[0] = 512; 
  //2K
  sizes[1] = 512;
  //4K
  sizes[2] = 64;
  //8K
  sizes[3] = 64;
  //16K
  sizes[4] = 32;
  //32K
  sizes[5] = 32;
  //64K
  sizes[6] = 32;
  //128K
  sizes[7] = 32;
  //256K
  sizes[8] = 32;
  //512K
  sizes[9] = 32;
  //1M
  sizes[10] = 170;
  //2M
  sizes[11] = 16;
  //4M
  sizes[12] = 4;
  //8M
  sizes[13] = 2;
  //16M
  sizes[14] = 2; 

  createPool(sizes, nslots, memPoolFreeBufs);
  printf("[%d] done creating buffer pool\n", CmiMyPe());

#endif

#ifdef GPU_INSTRUMENT_WRS
  initialized_instrument = false;
#endif
}

/* gpuProgressFn
 *  called periodically to monitor work request progress, and perform
 *  the prefetch of data for a subsequent work request
 */
void gpuProgressFn() {
  if (wrQueue == NULL) {
    printf("Error: work request queue not initialized\n"); 
    return; 
  }
  if (isEmpty(wrQueue)) {
    flushPinnedMemQueue();    
    flushDelayedFrees();
    return;
  } 
  int returnVal; 
  workRequest *head = firstElement(wrQueue); 
  workRequest *second = secondElement(wrQueue);
  workRequest *third = thirdElement(wrQueue); 

  if (head->state == QUEUED) {
#ifdef GPU_PROFILE
    gpuEvents[timeIndex].startTime = cutGetTimerValue(timerHandle); 
    gpuEvents[timeIndex].eventType = DATA_SETUP; 
    gpuEvents[timeIndex].ID = head->id; 
    dataSetupIndex = timeIndex; 
#ifdef GPU_TRACE
    gpuEvents[timeIndex].stage = GPU_MEM_SETUP; 
    gpuEvents[timeIndex].cmistartTime = CmiWallTimer();
#endif
    timeIndex++; 
#endif

#ifdef GPU_INSTRUMENT_WRS
    head->startTime = CmiWallTimer(); 
#endif

    allocateBuffers(head); 
    setupData(head); 
    head->state = TRANSFERRING_IN; 
  }  
  if (head->state == TRANSFERRING_IN) {
    if ((returnVal = hipStreamQuery(data_in_stream)) == hipSuccess) {
#ifdef GPU_PROFILE
      gpuEvents[dataSetupIndex].endTime = cutGetTimerValue(timerHandle);
#ifdef GPU_TRACE
      gpuEvents[dataSetupIndex].cmiendTime = CmiWallTimer();
      traceUserBracketEvent(gpuEvents[dataSetupIndex].stage, 
			    gpuEvents[dataSetupIndex].cmistartTime, 
			    gpuEvents[dataSetupIndex].cmiendTime); 
#endif 
#endif

#ifdef GPU_INSTRUMENT_WRS
      {
        if(initializedInstrument()){
          double tt = CmiWallTimer()-(head->startTime);
          int index = head->chareIndex;
          char type = head->compType;
          char phase = head->compPhase;

          CkVec<RequestTimeInfo> &vec = avgTimes[index][type];
          if(vec.length() <= phase){
            vec.growAtLeast(phase);
          }
          vec[phase].transferTime += tt;
        }
      }
#endif

      if (second != NULL /*&& (second->state == QUEUED)*/) {
	allocateBuffers(second); 
      }
#ifdef GPU_PROFILE
      gpuEvents[timeIndex].startTime = cutGetTimerValue(timerHandle); 
      gpuEvents[timeIndex].eventType = KERNEL_EXECUTION; 
      gpuEvents[timeIndex].ID = head->id; 
      runningKernelIndex = timeIndex; 
#ifdef GPU_TRACE
      gpuEvents[timeIndex].stage = GPU_KERNEL_EXEC; 
      gpuEvents[timeIndex].cmistartTime = CmiWallTimer();
#endif
      timeIndex++; 
#endif
#ifdef GPU_INSTRUMENT_WRS
      head->startTime = CmiWallTimer(); 
#endif

      //flushPinnedMemQueue();
      flushDelayedFrees();
      kernelSelect(head); 

      head->state = EXECUTING; 
      if (second != NULL) {
#ifdef GPU_PROFILE
	gpuEvents[timeIndex].startTime = cutGetTimerValue(timerHandle); 
	gpuEvents[timeIndex].eventType = DATA_SETUP; 
	gpuEvents[timeIndex].ID = second->id; 
	dataSetupIndex = timeIndex; 
#ifdef GPU_TRACE
	gpuEvents[timeIndex].stage = GPU_MEM_SETUP; 
	gpuEvents[timeIndex].cmistartTime = CmiWallTimer();
#endif
	timeIndex++; 
#endif

#ifdef GPU_INSTRUMENT_WRS
        second->startTime = CmiWallTimer();
#endif
	setupData(second); 
	second->state = TRANSFERRING_IN;
      }
    }
      /*
#ifdef GPU_DEBUG
      printf("Querying memory stream returned: %d %.2f\n", returnVal, 
	     cutGetTimerValue(timerHandle));
#endif  
      */
  }
  if (head->state == EXECUTING) {
    if ((returnVal = hipStreamQuery(kernel_stream)) == hipSuccess) {
#ifdef GPU_PROFILE
      gpuEvents[runningKernelIndex].endTime = cutGetTimerValue(timerHandle); 
#ifdef GPU_TRACE
      gpuEvents[runningKernelIndex].cmiendTime = CmiWallTimer();
      traceUserBracketEvent(gpuEvents[runningKernelIndex].stage, 
			    gpuEvents[runningKernelIndex].cmistartTime, 
			    gpuEvents[runningKernelIndex].cmiendTime); 
#endif
#endif
#ifdef GPU_INSTRUMENT_WRS
      {
        if(initializedInstrument()){
          double tt = CmiWallTimer()-(head->startTime);
          int index = head->chareIndex;
          char type = head->compType;
          char phase = head->compPhase;

          CkVec<RequestTimeInfo> &vec = avgTimes[index][type];
          if(vec.length() <= phase){
            vec.growAtLeast(phase);
          }
          vec[phase].kernelTime += tt;
        }
      }
#endif

      if (second != NULL && second->state == QUEUED) {
#ifdef GPU_PROFILE
	gpuEvents[timeIndex].startTime = cutGetTimerValue(timerHandle); 
	gpuEvents[timeIndex].eventType = DATA_SETUP; 
	gpuEvents[timeIndex].ID = second->id; 
	dataSetupIndex = timeIndex; 
#ifdef GPU_TRACE
	gpuEvents[timeIndex].stage = GPU_MEM_SETUP; 
	gpuEvents[timeIndex].cmistartTime = CmiWallTimer();
#endif
	timeIndex++; 
#endif

#ifdef GPU_INSTRUMENT_MS
        second->startTime = CmiWallTimer();
#endif
        
	allocateBuffers(second); 
	setupData(second); 
	second->state = TRANSFERRING_IN; 	
      } 
      if (second != NULL && second->state == TRANSFERRING_IN) {
	if (hipStreamQuery(data_in_stream) == hipSuccess) {
#ifdef GPU_PROFILE
	  gpuEvents[dataSetupIndex].endTime = cutGetTimerValue(timerHandle); 
#ifdef GPU_TRACE
	  gpuEvents[dataSetupIndex].cmiendTime = CmiWallTimer();
	  traceUserBracketEvent(gpuEvents[dataSetupIndex].stage, 
				gpuEvents[dataSetupIndex].cmistartTime, 
				gpuEvents[dataSetupIndex].cmiendTime); 
#endif
#endif
#ifdef GPU_INSTRUMENT_WRS
          {
            if(initializedInstrument()){
              double tt = CmiWallTimer()-(head->startTime);
              int index = second->chareIndex;
              char type = second->compType;
              char phase = second->compPhase;

              CkVec<RequestTimeInfo> &vec = avgTimes[index][type];
              if(vec.length() <= phase){
                vec.growAtLeast(phase);
              }
              vec[phase].transferTime += tt;
            }
          }
#endif

	  if (third != NULL /*&& (third->state == QUEUED)*/) {
	    allocateBuffers(third); 
	  }
#ifdef GPU_PROFILE
	  gpuEvents[timeIndex].startTime = cutGetTimerValue(timerHandle); 
	  gpuEvents[timeIndex].eventType = KERNEL_EXECUTION; 
	  gpuEvents[timeIndex].ID = second->id; 
	  runningKernelIndex = timeIndex; 
#ifdef GPU_TRACE
	  gpuEvents[timeIndex].stage = GPU_KERNEL_EXEC; 
	  gpuEvents[timeIndex].cmistartTime = CmiWallTimer();
#endif
	  timeIndex++; 
#endif
#ifdef GPU_INSTRUMENT_WRS
          second->startTime = CmiWallTimer();
#endif
	  //	    flushPinnedMemQueue();	    
          flushDelayedFrees();
	  kernelSelect(second); 
	  second->state = EXECUTING; 
	  if (third != NULL) {
#ifdef GPU_PROFILE
	    gpuEvents[timeIndex].startTime = cutGetTimerValue(timerHandle); 
	    gpuEvents[timeIndex].eventType = DATA_SETUP; 
	    gpuEvents[timeIndex].ID = third->id; 
	    dataSetupIndex = timeIndex; 
#ifdef GPU_TRACE
	    gpuEvents[timeIndex].stage = GPU_MEM_SETUP; 
	    gpuEvents[timeIndex].cmistartTime = CmiWallTimer();
#endif
	    timeIndex++; 
#endif

#ifdef GPU_INSTRUMENT_WRS
            third->startTime = CmiWallTimer();
#endif
	    setupData(third); 
	    third->state = TRANSFERRING_IN; 	
	  }
	}
      }
#ifdef GPU_PROFILE
      gpuEvents[timeIndex].startTime = cutGetTimerValue(timerHandle); 
      gpuEvents[timeIndex].eventType = DATA_CLEANUP; 
      gpuEvents[timeIndex].ID = head->id; 
      dataCleanupIndex = timeIndex; 	
#ifdef GPU_TRACE
      gpuEvents[timeIndex].stage = GPU_MEM_CLEANUP; 
      gpuEvents[timeIndex].cmistartTime = CmiWallTimer();
#endif
      timeIndex++; 
#endif
#ifdef GPU_INSTRUMENT_WRS
      head->startTime = CmiWallTimer(); 
#endif
      copybackData(head);
      head->state = TRANSFERRING_OUT;
    }
      /*
#ifdef GPU_DEBUG
      printf("Querying kernel completion returned: %d %.2f\n", returnVal,
	     cutGetTimerValue(timerHandle));
#endif  
      */
  }
  if (head->state == TRANSFERRING_OUT) {
    if (hipStreamQuery(data_out_stream) == hipSuccess && hipStreamQuery(kernel_stream) == hipSuccess){
      freeMemory(head); 
#ifdef GPU_PROFILE
      gpuEvents[dataCleanupIndex].endTime = cutGetTimerValue(timerHandle);
#ifdef GPU_TRACE
      gpuEvents[dataCleanupIndex].cmiendTime = CmiWallTimer();
      traceUserBracketEvent(gpuEvents[dataCleanupIndex].stage, 
			    gpuEvents[dataCleanupIndex].cmistartTime, 
			    gpuEvents[dataCleanupIndex].cmiendTime); 
#endif
#endif
#ifdef GPU_INSTRUMENT_WRS
      {
        if(initializedInstrument()){
          double tt = CmiWallTimer()-(head->startTime);
          int index = head->chareIndex;
          char type = head->compType;
          char phase = head->compPhase;

          CkVec<RequestTimeInfo> &vec = avgTimes[index][type];
          if(vec.length() <= phase){
            vec.growAtLeast(phase);
          }
          vec[phase].cleanupTime += tt;
          vec[phase].n++;
        }
      }
#endif

      dequeue(wrQueue);
      CUDACallbackManager(head->callbackFn);
    }
  }
}

#ifdef GPU_MEMPOOL
void releasePool(CkVec<BufferPool> &pools);
#endif
/* exitHybridAPI
 *  cleans up and deletes memory allocated for the queue and the CUDA streams
 */
void exitHybridAPI() {
  deleteWRqueue(wrQueue); 
  CUDA_SAFE_CALL_NO_SYNC(hipStreamDestroy(kernel_stream)); 
  CUDA_SAFE_CALL_NO_SYNC(hipStreamDestroy(data_in_stream)); 
  CUDA_SAFE_CALL_NO_SYNC(hipStreamDestroy(data_out_stream)); 

#ifdef GPU_PROFILE
  for (int i=0; i<timeIndex; i++) {
    switch (gpuEvents[i].eventType) {
    case DATA_SETUP:
      printf("Kernel %d data setup", gpuEvents[i].ID); 
      break;
    case DATA_CLEANUP:
      printf("Kernel %d data cleanup", gpuEvents[i].ID); 
      break; 
    case KERNEL_EXECUTION:
      printf("Kernel %d execution", gpuEvents[i].ID); 
      break;
    default:
      printf("Error, invalid timer identifier\n"); 
    }
    printf(" %.2f:%.2f\n", gpuEvents[i].startTime-gpuEvents[0].startTime, gpuEvents[i].endTime-gpuEvents[0].startTime); 
  }

  CUT_SAFE_CALL(cutStopTimer(timerHandle));
  CUT_SAFE_CALL(cutDeleteTimer(timerHandle));  

#endif

#ifdef GPU_MEMPOOL
  releasePool(memPoolFreeBufs);
#endif

}

#ifdef GPU_MEMPOOL
void releasePool(CkVec<BufferPool> &pools){
  for(int i = 0; i < pools.length(); i++){
    CUDA_SAFE_CALL_NO_SYNC(hipHostFree((void *)pools[i].head));
  }
  pools.free();
}

// Create a pool with nslots slots.
// There are nbuffers[i] buffers for each buffer size corresponding to entry i
// FIXME - list the alignment/fragmentation issues with either of two allocation schemes:
// if a single, large buffer is allocated for each subpool
// if multiple smaller buffers are allocated for each subpool
void createPool(int *nbuffers, int nslots, CkVec<BufferPool> &pools){
  //pools  = (BufferPool *)malloc(nslots*sizeof(BufferPool));
  pools.reserve(nslots);
  pools.length() = nslots;

  for(int i = 0; i < nslots; i++){
    int bufSize = memPoolBoundaries[i];
    int numBuffers = nbuffers[i];
    pools[i].size = bufSize;
    
    CUDA_SAFE_CALL_NO_SYNC(hipHostMalloc((void **)(&pools[i].head), 
                                          (sizeof(Header)+bufSize)*numBuffers));
    if(pools[i].head == NULL){
      abort();
    }

    Header *hd = pools[i].head;
    Header *previous = NULL;
    char *memory;

    for(int j = 0; j < numBuffers; j++){
      hd->slot = i;
      hd->next = previous;
      previous = hd;
      hd++; // move ptr past header
      memory = (char *)hd;
      memory += bufSize;
      hd = (Header *)memory;
    }

    pools[i].head = previous;
#ifdef GPU_MEMPOOL_DEBUG
    pools[i].num = numBuffers;
#endif
  }
}

int findPool(int size){
  int boundaryArrayLen = memPoolBoundaries.length();
  if(size <= memPoolBoundaries[0]){
    return (0);
  }
  else if(size > memPoolBoundaries[boundaryArrayLen-1]){
    // create new slot
    memPoolBoundaries.push_back(size);

    BufferPool newpool;
    CUDA_SAFE_CALL_NO_SYNC(hipHostMalloc((void **)&newpool.head, size+sizeof(Header)));
    newpool.size = size;
#ifdef GPU_MEMPOOL_DEBUG
    newpool.num = 1;
#endif
    memPoolFreeBufs.push_back(newpool);

    Header *hd = newpool.head;
    hd->next = NULL;
    hd->slot = boundaryArrayLen;

    return boundaryArrayLen;
  }
  for(int i = 0; i < GPU_MEMPOOL_NUM_SLOTS-1; i++){
    if(memPoolBoundaries[i] < size && size <= memPoolBoundaries[i+1]){
      return (i+1);
    }
  }
  return -1;
}

void *getBufferFromPool(int pool, int size){
  Header *ret;
  if(pool < 0 || pool >= memPoolFreeBufs.length() || memPoolFreeBufs[pool].head == NULL){
#ifdef GPU_MEMPOOL_DEBUG
    printf("(%d) pool %d size: %d, num: %d\n", CmiMyPe(), pool, size, memPoolFreeBufs[pool].num);
#endif
    abort();
  }
  else{
    ret = memPoolFreeBufs[pool].head;
    memPoolFreeBufs[pool].head = ret->next;
#ifdef GPU_MEMPOOL_DEBUG
    ret->size = size;
    memPoolFreeBufs[pool].num--;
#endif
    return (void *)(ret+1);
  }
  return NULL;
}

void returnBufferToPool(int pool, Header *hd){
  hd->next = memPoolFreeBufs[pool].head;
  memPoolFreeBufs[pool].head = hd;
#ifdef GPU_MEMPOOL_DEBUG
  memPoolFreeBufs[pool].num++;
#endif
}

void *hapi_poolMalloc(int size){
  int pool = findPool(size);
  void *buf = getBufferFromPool(pool, size);
#ifdef GPU_MEMPOOL_DEBUG
  printf("(%d) hapi_malloc size %d pool %d left %d\n", CmiMyPe(), size, pool, memPoolFreeBufs[pool].num);
#endif
  return buf;
}

void hapi_poolFree(void *ptr){
  Header *hd = ((Header *)ptr)-1;
  int pool = hd->slot;
  returnBufferToPool(pool, hd);
#ifdef GPU_MEMPOOL_DEBUG
  int size = hd->size;
  printf("(%d) hapi_free size %d pool %d left %d\n", CmiMyPe(), size, pool, memPoolFreeBufs[pool].num);
#endif
}


#endif

#ifdef GPU_INSTRUMENT_WRS
void hapi_initInstrument(int numChares, char types){
  avgTimes.reserve(numChares);
  avgTimes.length() = numChares;
  for(int i = 0; i < numChares; i++){
    avgTimes[i].reserve(types);
    avgTimes[i].length() = types;
  }
  initialized_instrument = true;
}

bool initializedInstrument(){
  return initialized_instrument;
}

RequestTimeInfo &hapi_queryInstrument(int chare, char type, char phase){
  return avgTimes[chare][type][phase];
}

void hapi_clearInstrument(){
  for(int chare = 0; chare < avgTimes.length(); chare++){
    for(int type = 0; type < avgTimes[chare].length(); type++){
      for(int phase = 0; phase < avgTimes[chare][type].length(); phase++){
        avgTimes[chare][type][phase].transferTime = 0.0;
        avgTimes[chare][type][phase].kernelTime = 0.0;
        avgTimes[chare][type][phase].cleanupTime = 0.0;
        avgTimes[chare][type][phase].n = 0;
      }
      avgTimes[chare][type].length() = 0;
    }
    avgTimes[chare].length() = 0;
  }
  avgTimes.length() = 0;
  initialized_instrument = false;
}
#endif
