#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hapi.h"

__global__ void helloKernel() {}

void runHello(struct workRequest* wr, hipStream_t kernel_stream,
              void** deviceBuffers) {
  helloKernel<<<wr->grid_dim, wr->block_dim, wr->shared_mem, kernel_stream>>>();
}

void kernelSetup(hipStream_t stream, void* cb) {
#ifdef USE_WR
  // DEPRECATED
  workRequest* wr = hapiCreateWorkRequest();
  wr->setExecParams(dim3(1, 1), dim3(1, 1));
  wr->setStream(stream);
  wr->setCallback(cb);
  wr->setTraceName("hello");
  wr->setRunKernel(runHello);

  hapiEnqueue(wr);
#else
  helloKernel<<<dim3(1, 1), dim3(1, 1), 0, stream>>>();
  hapiAddCallback(stream, cb);
#endif
}
